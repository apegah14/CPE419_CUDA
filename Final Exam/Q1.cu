
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(1234, tid, 0, &state[tid]);
}


__global__ void sampleCUDA(int *num_inside, hiprandState *states) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int num_inside_block[1024];

    num_inside_block[threadIdx.x] = 0;
    __syncthreads();
    float x = hiprand_uniform(&states[tid]);
    float y = hiprand_uniform(&states[tid]);
    //printf("thread: %d / x = %.3f / y = %.3f\n", tid, x, y);
    if (sqrt((x * x) + (y * y)) <= 1) {
        num_inside_block[threadIdx.x] = 1;
    }
    __syncthreads();

    for (int i = 1; i < blockDim.x; i *= 2) {
        int index = 2 * i * threadIdx.x;

        if (index < blockDim.x) {
            num_inside_block[index] += num_inside_block[index + i];
        }
        __syncthreads();
    }   

    if (threadIdx.x == 0) {
        num_inside[blockIdx.x] = num_inside_block[threadIdx.x];
        //printf("num_inside: %d\n", num_inside_block[threadIdx.x]);
    }
}

__global__ void reductionSum(int *num_inside, float *total) {
    __shared__ int reduction[1024];

    // bring in data from global memory to shared
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    reduction[threadIdx.x] = num_inside[tid];
    //printf("thread: %d / num_inside: %d\n", tid, reduction[threadIdx.x]);
    __syncthreads();
    for (int i = 1; i < blockDim.x; i *= 2) {
        int index = 2 * i * threadIdx.x;

        if (index < blockDim.x) {
            reduction[index] += reduction[index + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        total[0] = reduction[threadIdx.x];
        //printf("total: %d\n", reduction[threadIdx.x]);
    }
}

int main() {
    int samples = 1024 * 1000;
    int block_size = 1024;
    int grid_size = (samples + block_size - 1) / block_size;
    float *total_cpu = (float*)malloc(sizeof(float));

    int *num_inside_gpu;
    float *total;

    hipMalloc((void**)&num_inside_gpu, sizeof(int) * grid_size);
    hipMalloc((void**)&total, sizeof(float));

    // cuRand setup
    hiprandState_t *states;
    hipMalloc((void**) &states, samples * sizeof(hiprandState_t));
    setup_kernel<<<grid_size, block_size>>>(states);

    sampleCUDA<<<grid_size, block_size>>>(num_inside_gpu, states);

    reductionSum<<<1, 1024>>>(num_inside_gpu, total);
    hipMemcpy(total_cpu, total, sizeof(float), hipMemcpyDeviceToHost);

    float pi = (4 * total_cpu[0]) / samples;
    printf("Pi = %.6f\n", pi);

    printf("total = %.3f\n", total_cpu[0]);

    return 0;
}